/**
 * Copyright 2021 RICOS Co. Ltd.
 *
 * This file is a part of ricosjp/monolish,
 * and distributed under Apache-2.0 License
 * https://github.com/ricosjp/monolish
 */

#include "hip/hip_runtime.h"
#include <iostream>

int main(int argc, char **argv) {
  if (argc != 2) {
    std::cout << "Usage: " << argv[0] << " [device number]" << std::endl;
    return 1;
  }
  int device_number = std::stoi(argv[1]);

  hipError_t cudaStatus;

  int count;
  cudaStatus = hipGetDeviceCount(&count);
  if (cudaStatus != hipSuccess) {
    std::cerr << "CUDA API hipGetDeviceCount failed" << cudaStatus << std::endl;
    return cudaStatus;
  }

  if (device_number >= count) {
    std::cerr << "Input device_number is larger than the number of GPU ("
              << device_number << " >= " << count << ")" << std::endl;
    return 1;
  }

  hipDeviceProp_t prop;
  cudaStatus = hipGetDeviceProperties(&prop, device_number);
  if (cudaStatus != hipSuccess) {
    std::cerr << "CUDA API hipGetDeviceProperties failed" << std::endl;
    return cudaStatus;
  }

  std::cout << prop.major << prop.minor << std::endl;
  return 0;
}
